#include "hip/hip_runtime.h"
/* Brooke Husic and Jared Dunnmon 
 * Final project CME 253
 * Due Feb 17 2017
 */

#include <fstream>
#include <iostream>
#include <math.h>
#include <string>
#include <vector>
#include <chrono>

#include "./debug.h"

#ifdef DEBUG
#define CUDA_CALL(F)  if( (F) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__); exit(-1);} 
#define CUDA_CHECK()  if( (hipPeekAtLastError()) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);} 
#else
#define CUDA_CALL(F) (F)
#define CUDA_CHECK() 
#endif

#define MY_LIGAND_SIZE 51
typedef std::chrono::high_resolution_clock Clock;

void ProteinSetup(std::string protein_inputfile,
                  std::vector<int>& prot_atomnums,
                  std::vector<int>& prot_resnums,
                  std::vector<std::vector<double>>& prot_xyz_coords){
    std::ifstream f(protein_inputfile.c_str());
      if (f.is_open()) {
            std::string klass, code, resname, chain;
            int atomnum, resnum;
            double x, y, z, occ, temp;

            while (f >> klass >> atomnum >> code >> resname
                     >> chain >> resnum >> x >> y >> z
                     >> occ >> temp){

                std::vector<double> temp_coord;
                temp_coord.push_back(x);
                temp_coord.push_back(y);
                temp_coord.push_back(z); 

                prot_atomnums.push_back(atomnum);
                prot_resnums.push_back(resnum);
                prot_xyz_coords.push_back(temp_coord);
            }

            // some checks
            if(prot_atomnums.size() != prot_resnums.size()){
                std::cerr << "ERROR: Problem in protein file" << std::endl;
            }

            if(prot_atomnums.size() != prot_xyz_coords.size()){
                std::cerr << "ERROR: Problem in protein file" << std::endl;
            }
    }
    std::cout << "Lines in protein file : " << prot_atomnums.size() << std::endl;
}

void LigandTrajSetup(std::string ligand_inputfile,
                     std::vector<int>& lig_trajnums,
                     std::vector<int>& lig_atomnums,
                     std::vector<int>& lig_resnums,
                     std::vector<std::vector<double>>& lig_xyz_coords){
    std::ifstream f(ligand_inputfile.c_str());
    if (f.is_open()) {
        std::string klass, code, resname, chain;
        int trajnum, atomnum, resnum;
        double x, y, z, occ, temp;

        while (f >> trajnum
                 >> klass >> atomnum >> code >> resname
                 >> chain >> resnum >> x >> y >> z
                 >> occ >> temp){

            std::vector<double> temp_coord;
            temp_coord.push_back(x);
            temp_coord.push_back(y);
            temp_coord.push_back(z); 

            lig_trajnums.push_back(trajnum);
            lig_atomnums.push_back(atomnum);
            lig_resnums.push_back(resnum);
            lig_xyz_coords.push_back(temp_coord);
        }

        // some checks
        if(lig_atomnums.size() != lig_trajnums.size()){
            std::cerr << "ERROR: Problem in ligand file" << std::endl;
        }
        if(lig_atomnums.size() != lig_resnums.size()){
            std::cerr << "ERROR: Problem in ligand file" << std::endl;
        }
        if(lig_atomnums.size() != lig_xyz_coords.size()){
            std::cerr << "ERROR: Problem in ligand file" << std::endl;
        }
    }
    std::cout << "Lines in ligand file : " << lig_atomnums.size() << std::endl;
    std::cout << "Ligand poses in file : " << lig_atomnums.size()/17 << std::endl;
}


double ComputeSquaredDistance(std::vector<double> v1, std::vector<double> v2){
    double dist_squared;
    dist_squared = {  (v1[0]-v2[0])*(v1[0]-v2[0])
                    + (v1[1]-v2[1])*(v1[1]-v2[1])
                    + (v1[2]-v2[2])*(v1[2]-v2[2]) };
    return dist_squared;
}

std::vector<double> LPContactFeaturizer(std::vector<int>& prot_atomnums,
                                        std::vector<std::vector<double>>& prot_xyz_coords,
                                        std::vector<int>& lig_trajnums,
                                        std::vector<std::vector<double>>& lig_xyz_coords){

    std::vector<double> all_distances;
    for (unsigned int ii = 0; ii < lig_trajnums.size(); ii++){
        for (unsigned int jj =0; jj < prot_atomnums.size(); jj++){ 
            double temp_dist = ComputeSquaredDistance(lig_xyz_coords[ii],
                                                      prot_xyz_coords[jj]);
            temp_dist = sqrt(temp_dist)/10.;
            all_distances.push_back(temp_dist);
        }
    }
    return all_distances;
}


__global__ void cuContacts(double *pxyz, double *lxyz, double *cudists, int *plength, int *llength)
{
  int pidx = threadIdx.x + blockIdx.x * blockDim.x;
  int lidx = threadIdx.y + blockIdx.y * blockDim.y;

  if ( (pidx < plength[0]) && (lidx< llength[0])){
    cudists[pidx+plength[0]*lidx] = ( sqrt(
               (pxyz[pidx*3]-lxyz[lidx*3])*(pxyz[pidx*3]-lxyz[lidx*3])
             + (pxyz[pidx*3+1]-lxyz[lidx*3+1])*(pxyz[pidx*3+1]-lxyz[lidx*3+1])
             + (pxyz[pidx*3+2]-lxyz[lidx*3+2])*(pxyz[pidx*3+2]-lxyz[lidx*3+2])  )/10. );

  }  
  __syncthreads();

}


__global__ void cuContactsSMEM(double *pxyz, double *lxyz, double *cudists, int *plength, int *llength)
{
  int pidx = threadIdx.x + blockIdx.x * blockDim.x;
  int lidx = threadIdx.y + blockIdx.y * blockDim.y;
  
  __shared__ double temp[MY_LIGAND_SIZE*3+1][2];
  if(lidx<llength[0]){
      temp[3*lidx][0] = lxyz[3*lidx];
      temp[3*lidx+1][0] = lxyz[3*lidx+1];
      temp[3*lidx+2][0] = lxyz[3*lidx+2];
}
  __syncthreads();

  if ( (pidx < plength[0]) && (lidx< llength[0])){
    cudists[pidx+plength[0]*lidx] = ( sqrt(
               (pxyz[pidx*3]-temp[lidx*3][0])*(pxyz[pidx*3]-temp[lidx*3][0])
             + (pxyz[pidx*3+1]-temp[lidx*3+1][0])*(pxyz[pidx*3+1]-temp[lidx*3+1][0])
             + (pxyz[pidx*3+2]-temp[lidx*3+2][0])*(pxyz[pidx*3+2]-temp[lidx*3+2][0])  )/10. );

  }  
  __syncthreads();

}


#define THREADS_PER_BLOCK_X 32
#define THREADS_PER_BLOCK_Y 32

int main(int argc, char *argv[])
{
  if (argc != 3)
  {
    std::cout << "Usage:" << std::endl;
    {std::cout << "  " << argv[0] << " <protein input file> "
                       << " <ligand input file> " << std::endl;}
    return 0;
  }

  std::string protein_inputfile = argv[1];
  std::string ligand_inputfile = argv[2];

  std::vector<int> prot_atomnums;
  std::vector<int> prot_resnums;
  std::vector<std::vector<double>> prot_xyz_coords;

  std::vector<int> lig_trajnums;
  std::vector<int> lig_atomnums;
  std::vector<int> lig_resnums;
  std::vector<std::vector<double>> lig_xyz_coords;

  ProteinSetup(protein_inputfile,
               prot_atomnums,
               prot_resnums,
               prot_xyz_coords);

  LigandTrajSetup(ligand_inputfile,
                  lig_trajnums,
                  lig_atomnums,
                  lig_resnums,
                  lig_xyz_coords);

  auto cpp_start = Clock::now();

  /* compute distanes using cpp*/
  std::vector<double> distances = LPContactFeaturizer(prot_atomnums,
                                                      prot_xyz_coords,
                                                      lig_trajnums,
                                                      lig_xyz_coords);

  auto cpp_end = Clock::now();

  /* print out cpp distances to a file */
  std::cout << "Number of distances to compute : " << distances.size() << std::endl;
  std::cout << "Cpp distances calculated in " 
              << std::chrono::duration_cast<std::chrono::microseconds>(cpp_end - cpp_start).count()
              << " microseconds" << std::endl;


//  std::cout << "First cpp distance computed : " << distances[0] << std::endl;
//  if(prot_atomnums.size()*lig_trajnums.size() > 50){
//    std::cout << "50th cpp distance computed : " << distances[50] << std::endl; }
//  if(prot_atomnums.size()*lig_trajnums.size() > 100000){
//    std::cout << "100kth cpp distance computed : " << distances[100000] << std::endl; }
  
  double *pxyz, *lxyz, *cudists;
  double *d_pxyz, *d_lxyz, *d_cudists;
  int *plength, *d_plength;
  int *llength, *d_llength;
  int protein_size = prot_atomnums.size()*3;
  int ligand_traj_size = lig_trajnums.size()*3;
  int cudists_size = protein_size/3 * ligand_traj_size/3;

/* get GPU device number and name */
  int dev;
  hipDeviceProp_t deviceProp;
  checkCUDA( hipGetDevice( &dev ) );
  checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );

/* allocate space for device copies of a, b, c */
  checkCUDA( hipMalloc( (void **) &d_pxyz, protein_size*sizeof(double)) );
  checkCUDA( hipMalloc( (void **) &d_lxyz, ligand_traj_size*sizeof(double)) );
  checkCUDA( hipMalloc( (void **) &d_cudists, cudists_size*sizeof(double) ));
  checkCUDA( hipMalloc( (void **) &d_plength, sizeof(int) ));
  checkCUDA( hipMalloc( (void **) &d_llength, sizeof(int) ));

/* allocate space for host copies of a, b, c and setup input values */
  pxyz = (double *)malloc( protein_size *sizeof(double));
  lxyz = (double *)malloc( ligand_traj_size *sizeof(double));
  cudists = (double *)malloc( cudists_size *sizeof(double));
  plength = (int *)malloc( sizeof(int));
  llength = (int *)malloc( sizeof(int));

  for(unsigned int pp = 0; pp < prot_atomnums.size(); pp++){
    pxyz[pp*3] = prot_xyz_coords[pp][0];
    pxyz[pp*3+1] = prot_xyz_coords[pp][1];
    pxyz[pp*3+2] = prot_xyz_coords[pp][2];
  }

  for(unsigned int ll = 0; ll < lig_trajnums.size(); ll++){
    lxyz[ll*3] = lig_xyz_coords[ll][0];
    lxyz[ll*3+1] = lig_xyz_coords[ll][1];
    lxyz[ll*3+2] = lig_xyz_coords[ll][2];
  }

  plength[0] = prot_atomnums.size();
  llength[0] = lig_trajnums.size();

/* copy inputs to device */
  checkCUDA( hipMemcpy( d_pxyz, pxyz, protein_size*sizeof(double), hipMemcpyHostToDevice ) );
  checkCUDA( hipMemcpy( d_lxyz, lxyz, ligand_traj_size*sizeof(double), hipMemcpyHostToDevice ) );
  checkCUDA( hipMemcpy( d_plength, plength, sizeof(int), hipMemcpyHostToDevice) );  
  checkCUDA( hipMemcpy( d_llength, llength, sizeof(int), hipMemcpyHostToDevice) );


/* zero out the C array */
  checkCUDA( hipMemset( d_cudists, 0, cudists_size*sizeof(double) ) );

/* setup threadblock size and grid sizes*/
  dim3 threads(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_X, 1);
  dim3 blocks(cudists_size/threads.x+1,
              cudists_size/threads.y+1,
              1 );

    hipDeviceProp_t prop;

    hipGetDeviceProperties(&prop, 0);
    if (threads.x * threads.y * threads.z > prop.maxThreadsPerBlock) {
        printf("Too many threads per block \n");
    }
    if (threads.x > prop.maxThreadsDim[0]) {
        printf("Too many threads in x-direction \n");
    }
    if (threads.y > prop.maxThreadsDim[1]) {
        printf("Too many threads in y-direction \n");
    }
    if (threads.z > prop.maxThreadsDim[2]) {
        printf("Too many threads in z-direction \n");
    }
   printf("Ready to launch kernel\n");

  auto cuda_start = Clock::now();

/* launch the kernel on the GPU */
  cuContactsSMEM<<< blocks, threads >>>( d_pxyz, d_lxyz, d_cudists, d_plength, d_llength );
  checkKERNEL();

  auto cuda_mid = Clock::now();

  /* print out cpp distances to a file */
//  std::cout << "Number of CUDA distances computed : " << distances.size() << std::endl;
  std::cout << "CUDA distances calculated in " 
              << std::chrono::duration_cast<std::chrono::microseconds>(cuda_mid - cuda_start).count()
              << " microseconds" << std::endl;

/* copy result back to host */
  checkCUDA( hipMemcpy( cudists, d_cudists, cudists_size*sizeof(double), hipMemcpyDeviceToHost ) );

  auto cuda_end = Clock::now();

  std::cout << "CUDA distances copied in "
              << std::chrono::duration_cast<std::chrono::microseconds>(cuda_end - cuda_mid).count()
              << " microseconds" << std::endl;

  std::cout << "Total: "
              << std::chrono::duration_cast<std::chrono::microseconds>(cuda_end - cuda_start).count()
              << " microseconds" << std::endl;

  //std::cout << "Number of cuda distances computed : " << sizeof cudists << std::endl; 
//  std::cout << "First cuda distance computed : " << cudists[0] << std::endl;
  // if(prot_atomnums.size()*lig_trajnums.size() > 50){
  //   std::cout << "50th cuda distance computed : " << cudists[50] << std::endl; }
  // if(prot_atomnums.size()*lig_trajnums.size() > 100000){
  //  std::cout << "100kth cuda distance computed : " << cudists[100000] << std::endl; }


  std::ofstream f("distances.txt");
  if(f.is_open()){
    for(unsigned int k = 0; k < distances.size(); k++){
                        f << distances[k] << "  " << cudists[k] << std::endl;
                    }
  }
  f.close();

  free(pxyz);
  free(lxyz);
  free(cudists);
  free(plength);
  checkCUDA( hipFree( d_pxyz ) );
  checkCUDA( hipFree( d_lxyz ) );
  checkCUDA( hipFree( d_cudists ) );
  checkCUDA( hipFree( d_plength ) );

  checkCUDA( hipDeviceReset () ); 
    
  return 0;
} /* end main */
